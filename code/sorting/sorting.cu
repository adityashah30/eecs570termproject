#include "hip/hip_runtime.h"
#include "sorting.cuh"
#include <sstream>

__device__
bool compFoo(Record* rec1, Record* rec2, int fieldIdx)
{
    switch(fieldIdx)
    {
        case 0:
            return rec1->userId < rec2->userId;
        case 1:
            return rec1->movieId < rec2->movieId;
        case 2:
            return rec1->rating < rec2->rating;
        case 3:
            return rec1->timestamp < rec2->timestamp;
        default:
            return rec1->rating < rec2->rating;
    }
}

__global__
void sortDataKernel(Record* out,
                    int fieldIdx,
                    size_t size)
{
    int blockId = blockIdx.x
                  + blockIdx.y*gridDim.x
                  + blockIdx.z*gridDim.y*gridDim.x;
    int threadId = threadIdx.x
                   + threadIdx.y*blockDim.x
                   + threadIdx.z*blockDim.x*blockDim.y
                   + blockId*blockDim.x*blockDim.y*blockDim.z;
    int numThreads = blockDim.x*blockDim.y*blockDim.z
                        *gridDim.x*gridDim.y*gridDim.z;

    int numComp = ((size>>1) + numThreads-1)/numThreads;
    
    for(int ostep = 2; ostep <= size; ostep <<= 1)
    {
        int halfStep = ostep >> 1;
        for(int istep = ostep; istep > 1; istep >>= 1)
        {
            int stride = istep >> 1;
            for(int i=0; i<numComp; i++)
            {
                int compId = threadId*numComp + i;
                if(compId >= size>>1)
                {
                    break;
                }
                int idx1 = (compId/stride)*istep + (compId%stride);
                int idx2 = idx1 + stride;
                Record* it1 = out + idx1;
                Record* it2 = out + idx2;
                bool dir = (compId%ostep) < halfStep;
                if(dir)
                {
                    if(compFoo(it2, it1, fieldIdx))
                    {
                        Record temp = *it1;
                        *it1 = *it2;
                        *it2 = temp;
                    }
                }
                else
                {
                    if(compFoo(it1, it2, fieldIdx))
                    {
                        Record temp = *it1;
                        *it1 = *it2;
                        *it2 = temp;
                    }
                }
            }
            __syncthreads();
        }
    }
}

void sortData(Dataset& out, Dataset& in, int index, int numThreads)
{
    Dataset_d out_d = in;
    size_t size = in.size();

    dim3 block(numThreads,1,1);

    Record* out_begin = thrust::raw_pointer_cast(out_d.data());

    sortDataKernel<<<1,block>>>(out_begin, index, size);
    checkCudaErrorKernel("sortDataKernel");

    out.resize(size);
    thrust::copy(out_d.begin(), out_d.end(), out.begin());
}

// __global__
// void sortDataKernel(Record* out,
//                     int ostep, int istep,
//                     int fieldIdx,
//                     size_t size,
//                     size_t numComp)
// {
//     size_t blockId = blockIdx.x
//                   + blockIdx.y*gridDim.x
//                   + blockIdx.z*gridDim.y*gridDim.x;
//     size_t threadId = threadIdx.x
//                    + threadIdx.y*blockDim.x
//                    + threadIdx.z*blockDim.x*blockDim.y
//                    + blockId*blockDim.x*blockDim.y*blockDim.z;
//     // int blockId = blockIdx.x;
//     // int threadId = blockIdx.x*blockDim.x + threadIdx.x;

//     // size_t numThreads = blockDim.x*blockDim.y*blockDim.z*
//     //                     gridDim.x*gridDim.y*gridDim.z;

//     // size_t numComp = (size+2*numThreads-1)/(2*numThreads);

//     int halfStep = ostep >> 1;
//     int stride = istep >> 1;
    
//     for(int i=0; i<numComp; i++)
//     {
//         int compId = threadId*numComp + i;

//         if(compId >= size>>1)
//         {
//             break;
//         }

//         int idx1 = (compId/stride)*istep + (compId%stride);
//         int idx2 = idx1 + stride;
//         Record* it1 = out + idx1;
//         Record* it2 = out + idx2;

//         printf("%d %d; %d %d %d; %x %x\n", blockId, threadId, compId, idx1, idx2, it1, it2);

//         bool dir = (compId%ostep) < halfStep;
//         if(dir)
//         {
//             if(compFoo(it2, it1, fieldIdx))
//             {
//                 thrust::swap(*it1, *it2);
//             }
//         }
//         else
//         {
//             if(compFoo(it1, it2, fieldIdx))
//             {
//                 thrust::swap(*it1, *it2);
//             }
//         }
//     }
// }

// void sortData(Dataset& out, Dataset& in, int index, int numThreads)
// {
//     Dataset_d out_d = in;
//     // checkCudaErrorKernel("Allocating device_vector");
//     size_t size = in.size();

//     dim3 block(64,1,1);
//     dim3 grid(1,1,1);

//     size_t numThreads1 = block.x*block.y*block.z*grid.x*grid.y*grid.z;

//     size_t numComp = (size+2*numThreads1-1)/(2*numThreads1);

//     std::cout << size << " " << numThreads1 << " " << numComp << std::endl;

//     Record* out_begin = thrust::raw_pointer_cast(out_d.data());

//     for(int ostep = 2; ostep <= size; ostep <<= 1)
//     {
//         for(int istep = ostep; istep > 1; istep >>= 1)
//         {
//             sortDataKernel<<<grid,block>>>(out_begin, ostep, istep, index, size, numComp);
//             std::stringstream s;
//             s << "sortDataKernel " << ostep << ", " << istep;
//             checkCudaErrorKernel(s.str());
//         }
//     }

//     out.resize(in.size());
//     thrust::copy(out_d.begin(), out_d.end(), out.begin());
//     checkCudaErrorKernel("Copying from deivce to host");
// }
