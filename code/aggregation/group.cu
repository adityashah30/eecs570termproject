#include "hip/hip_runtime.h"
#include "group.cuh"
#include <sstream>

#define ELEMENTS 26744
#define HASH_ENTRIES 1024

__device__ __host__ 
int hash(int value, size_t count) {
	return value % count;
}

__global__ 
void aggre_kernel(Entry** entries, Entry* pool, int* first_free, size_t size, Record* in){
    size_t blockId = blockIdx.x
                  + blockIdx.y*gridDim.x
                  + blockIdx.z*gridDim.y*gridDim.x;
    size_t threadId = threadIdx.x
                   + threadIdx.y*blockDim.x
                   + threadIdx.z*blockDim.x*blockDim.y
                   + blockId*blockDim.x*blockDim.y*blockDim.z;
    size_t numThreads = blockDim.x*blockDim.y*blockDim.z
                        *gridDim.x*gridDim.y*gridDim.z;

	if(threadId < numThreads){
		int chunksize = (size + numThreads-1)/numThreads;
	
		Entry** mytable = entries + HASH_ENTRIES * threadId;
		int* my_first_free = first_free;
		Entry* mypool = pool + ELEMENTS * threadId;
	
		assert(mypool[0].key == -1);
		for(int i = 0; i < chunksize; ++i){
			int cur_id = threadId * chunksize + i;
			int key = in[cur_id].movieId;
			int hashval = hash(key, HASH_ENTRIES);
		
			Entry *cur = mytable[hashval];	
			while(cur){
				if(cur->key == key){
					cur->value += in[cur_id].rating;
					++(cur->cnt);
					break;
				}
				cur = cur->next;
			}
			if(!cur){
				Entry *new_entry = &(mypool[*my_first_free]);
				(*my_first_free)++;
				new_entry->key = key;
				new_entry->value = in[cur_id].rating;
				new_entry->cnt = 1;
				new_entry->next = mytable[hashval];
				mytable[hashval] = new_entry;
			}
		}
	}
	__syncthreads();
}


void group(Dataset& out, Dataset& in, int numThreads){
    Dataset_d in_d = in;
	size_t size = in.size();

	int tpb = (numThreads<1024)?numThreads:1024;
    dim3 block(tpb,1,1);
	int numBlocks = (numThreads+tpb-1)/tpb;
	dim3 grid(numBlocks,1,1);
	

	//int totalThreads = numThreads * block_1;
	//Tables_d tables_d(numThreads);
	
	//for(int i = 0; i < numThreads; ++i)
	//	initialize_table(tables_d[i], HASH_ENTRIES, ELEMENTS);
	Entries_ptr_d total_entries_d(HASH_ENTRIES * numThreads);
	Entries_d total_pool_d(ELEMENTS * numThreads);
	vecint_d first_free_d(numThreads);
	
	Record* in_begin = thrust::raw_pointer_cast(in_d.data());
	//Table* table_begin = thrust::raw_pointer_cast(tables_d.data());
	Entry** entries_begin = thrust::raw_pointer_cast(total_entries_d.data());
	Entry*  pool_begin = thrust::raw_pointer_cast(total_pool_d.data());
	int* first_free_begin = thrust::raw_pointer_cast(first_free_d.data());
	
	aggre_kernel<<<grid, block>>>(entries_begin, pool_begin, first_free_begin, size, in_begin);
	
	std::vector<Entry> host_pools(ELEMENTS * numThreads);
	
	thrust::copy(total_pool_d.begin(), total_pool_d.end(), host_pools.begin());

	std::vector<Entry*> final_entries;
	std::vector<Entry> final_pool;
	int final_first_free = 0;
	
	// Build the table at host
	for(int i = 0; i < numThreads; ++i){
		int start = i * ELEMENTS;
		int end = start + ELEMENTS;
		for(int j = start; j < end; ++j){
			if(host_pools[j].key == -1)
				break;
			
			int key = host_pools[j].key;
			int hashval = hash(key, HASH_ENTRIES);
		
			Entry *cur = final_entries[hashval];
			while(cur){
				if(cur->key == key){
					cur->value += host_pools[j].value;
					cur->cnt   += host_pools[j].cnt;
				break;
				}
				cur = cur->next;
			}
			if(!cur){
				Entry *new_entry = &final_pool[final_first_free++];
				new_entry->key = key;
				new_entry->value = host_pools[j].value;
				new_entry->cnt   = host_pools[j].cnt;
				new_entry->next = final_entries[hashval];
				final_entries[hashval] = new_entry;
			}
		}
	}
	
	// Traverse the table, calculate the avg
	assert(final_first_free == ELEMENTS);
	out.resize(size);
	
	int k = 0;
	for(int i = 0; i < HASH_ENTRIES; ++i){
		Entry *cur = final_entries[i];
		while(cur){
			double avg_rating = cur->value / cur->cnt;
			Record record(-1, cur->key, avg_rating, -1);
			out[k++] = record;
		}
	}
	
}