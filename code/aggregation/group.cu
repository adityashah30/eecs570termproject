#include "hip/hip_runtime.h"
#include "group.cuh"
#include <sstream>
//#include "cuPrintf.cu"

#define ELEMENTS 16384
#define HASH_ENTRIES 512  

//using namespace std;

__device__ __host__ 
int hash(int value, size_t count) {
	return value % count;
}

__global__ 
void aggre_kernel(Entry** entries, Entry* pool, int* first_free, size_t size, Record* in){
    size_t blockId = blockIdx.x
                  + blockIdx.y*gridDim.x
                  + blockIdx.z*gridDim.y*gridDim.x;
    size_t threadId = threadIdx.x
                   + threadIdx.y*blockDim.x
                   + threadIdx.z*blockDim.x*blockDim.y
                   + blockId*blockDim.x*blockDim.y*blockDim.z;
    size_t numThreads = blockDim.x*blockDim.y*blockDim.z
                        *gridDim.x*gridDim.y*gridDim.z;

	if(threadId < numThreads){
		int chunksize = (size + numThreads-1)/numThreads;
		//std::cout << "thread id: " << threadID << std::endl;	
		//std::cout << "chunk size: " << chunksize << std::endl;
	//	printf("Thread id: %d\n", threadId);
	//	printf("Chunk size: %d\n", chunksize);
		Entry** mytable = entries + HASH_ENTRIES * threadId;
		int* my_first_free = first_free + threadId;
		Entry* mypool = pool + ELEMENTS * threadId;
	
		//assert(mypool[0].key == -1);
		int idx1 = threadId * chunksize;
     		int idx2 = (threadId + 1) * chunksize;
        	if(threadId == numThreads-1){
        	    idx2 = size;
        	}
		for(int cur_id = idx1; cur_id < idx2; ++cur_id){
			//int cur_id = threadId * chunksize + i;
			int key = in[cur_id].movieId;
			int hashval = hash(key, HASH_ENTRIES);
		
			Entry *cur = mytable[hashval];	
			while(cur){
				if(cur->key == key){
					cur->value += in[cur_id].rating;
					++(cur->cnt);
					break;
				}
				cur = cur->next;
			}
			if(!cur){
				Entry *new_entry = &(mypool[*my_first_free]);
				(*my_first_free)++;
				new_entry->key = key;
				new_entry->value = in[cur_id].rating;
				new_entry->cnt = 1;
				new_entry->next = mytable[hashval];
				mytable[hashval] = new_entry;
			}
		}
	}
	__syncthreads();
}


void group(Dataset& out, Dataset& in, int numThreads){
    	Dataset_d in_d = in;
	out.clear();
	size_t size = in.size();

	int tpb = (numThreads<1024)?numThreads:1024;
    dim3 block(tpb,1,1);
	int numBlocks = (numThreads+tpb-1)/tpb;
	dim3 grid(numBlocks,1,1);
	

	//int totalThreads = numThreads * block_1;
	//Tables_d tables_d(numThreads);
	
	//for(int i = 0; i < numThreads; ++i)
	//	initialize_table(tables_d[i], HASH_ENTRIES, ELEMENTS);
	std::vector<Entry> final_pool(ELEMENTS * numThreads);

	Entries_ptr_d total_entries_d(HASH_ENTRIES * numThreads, NULL);
	Entries_d total_pool_d = final_pool;
	vecint_d first_free_d(numThreads, 0);
	
	Record* in_begin = thrust::raw_pointer_cast(in_d.data());
	//Table* table_begin = thrust::raw_pointer_cast(tables_d.data());
	Entry** entries_begin = thrust::raw_pointer_cast(total_entries_d.data());
	Entry*  pool_begin = thrust::raw_pointer_cast(total_pool_d.data());
	int* first_free_begin = thrust::raw_pointer_cast(first_free_d.data());

	std::cout << "start aggregation kernel" << std::endl;	
	aggre_kernel<<<grid, block>>>(entries_begin, pool_begin, first_free_begin, size, in_begin);
	checkCudaErrorKernel("aggregationKernel");
	std::cout << "end aggregation kernel" << std::endl;
	
	std::vector<Entry> host_pools(ELEMENTS * numThreads);
	
	thrust::copy(total_pool_d.begin(), total_pool_d.end(), host_pools.begin());

	std::vector<Entry*> final_entries(HASH_ENTRIES, NULL);
//	std::vector<Entry> final_pool;
	int final_first_free = 0;
	
	// Build the table at host
	std::cout << "start build table at host" << std::endl;
	for(int i = 0; i < numThreads; ++i){
		//std::cout << "start building table on thread id: " << i << std::endl;
		int start = i * ELEMENTS;
		int end = start + ELEMENTS;
		for(int j = start; j < end; ++j){
			//std::cout << "key value: " << host_pools[j].key << std::endl;
			if(host_pools[j].key == -1)
				break;
			
			int key = host_pools[j].key;
			int hashval = hash(key, HASH_ENTRIES);
			//std::cout << "hash value: " << hashval << std::endl;
			Entry *cur = final_entries[hashval];
			while(cur){
				if(cur->key == key){
					cur->value += host_pools[j].value;
					cur->cnt   += host_pools[j].cnt;
				break;
				}
				cur = cur->next;
			}
			if(!cur){
				Entry *new_entry = &final_pool[final_first_free++];
				new_entry->key = key;
				new_entry->value = host_pools[j].value;
				new_entry->cnt   = host_pools[j].cnt;
				new_entry->next = final_entries[hashval];
				final_entries[hashval] = new_entry;
			}
		}
	}
	
	// Traverse the table, calculate the avg
	std::cout << "final first free: " << final_first_free << std::endl;
	assert(final_first_free == ELEMENTS);
	std::cout << "finish build table at host" << std::endl;
	out.resize(ELEMENTS);
	
	int k = 0;
	for(int i = 0; i < HASH_ENTRIES; ++i){
		Entry *cur = final_entries[i];
		while(cur){
			double avg_rating = cur->value / cur->cnt;
			Record record(-1, cur->key, avg_rating, -1);
			out[k++] = record;
			cur = cur->next;
		}
	}
	std::cout << "complete populate to out" << std::endl;
	
}
